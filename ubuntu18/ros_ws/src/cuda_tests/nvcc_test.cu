//nvidia-toolkit need to be installed
//run^
//RUN: nvcc nvcc_test.cu -o test && ./nvcc_test 

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Hello from GPU thread %d!\n", threadIdx.x);
}

int main() {
    kernel<<<1, 5>>>();
    hipDeviceSynchronize();
    return 0;
}